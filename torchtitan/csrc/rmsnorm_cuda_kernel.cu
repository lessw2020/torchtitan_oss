#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
#include "ATen/AccumulateType.h"
#include "ATen/cuda/HIPContext.h"
#include "ATen/cuda/DeviceUtils.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "type_shim.h"
#include "static_switch.h"


template<typename AccType>
__device__ void cuRMSOnlineSum(const AccType curr, AccType& sigma2) {
  sigma2 = sigma2 + curr * curr;
}


template<typename AccType>
__device__ void cuChanRMSOnlineSum(const AccType sigma2B, AccType& sigma2) {
  sigma2 = sigma2 + sigma2B;
}

// updated to remove Mu since we only care about RMSNorm
template<typename InputType, typename AccType>
__device__ void cuWelfordSigma2(
    const InputType* __restrict__ vals, const int n1, const int n2, const int i1,
    AccType& sigma2, AccType* buf) {
    // Assumptions:
    // 1) blockDim.x == warpSize
    // 2) Tensor is contiguous
    // 3) blockDim.y*sizeof(AccType) shared memory available.
    //
    // compute sum of squares over n2
    sigma2 = AccType(0);
    if (i1 < n1) {
        // one warp normalizes one n1 index,
        // synchronization is implicit
        const int numx = blockDim.x * blockDim.y;
        const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
        const InputType* lvals = vals + i1*n2;
        int l = 4*thrx;
        for (; l+3 < n2; l+=4*numx) {
            for (int k = 0; k < 4; ++k) {
                AccType curr = static_cast<AccType>(__ldg(&lvals[l+k]));
                cuRMSOnlineSum<AccType>(curr, sigma2);
            }
        }
        for (; l < n2; ++l) {
            AccType curr = static_cast<AccType>(__ldg(&lvals[l]));
            cuRMSOnlineSum<AccType>(curr, sigma2);
        }
        // intra-warp reductions
        #pragma unroll
        for (int l = 0; l <= 4; ++l) {
            int srcLaneB = (threadIdx.x+(1<<l))&31;
            AccType sigma2B = WARP_SHFL(sigma2, srcLaneB);
            cuChanRMSOnlineSum<AccType>(sigma2B, sigma2);
        }
        // threadIdx.x == 0 has correct values for each warp
        // inter-warp reductions
        if (blockDim.y > 1) {
            AccType* ubuf = static_cast<AccType*>(buf);
            for (int offset = blockDim.y/2; offset > 0; offset /= 2) {
                // upper half of warps write to shared
                if (threadIdx.x == 0 && threadIdx.y >= offset && threadIdx.y < 2*offset) {
                    const int wrt_y = threadIdx.y - offset;
                    ubuf[wrt_y] = sigma2;
                }
                __syncthreads();
                // lower half merges
                if (threadIdx.x == 0 && threadIdx.y < offset) {
                    AccType sigma2B = ubuf[threadIdx.y];
                    cuChanRMSOnlineSum<AccType>(sigma2B, sigma2);
                }
                __syncthreads();
            }
            // threadIdx.x = 0 && threadIdx.y == 0 only thread that has correct value
            if (threadIdx.x == 0 && threadIdx.y == 0) {
                ubuf[0] = sigma2;
            }
            __syncthreads();
            sigma2 = ubuf[0] / AccType(n2);
        } else {
            sigma2 = WARP_SHFL(sigma2 / AccType(n2), 0);
        }
    }
}

// rsqrts

template<typename AccType>
__device__ __forceinline__ AccType rsqrt(AccType v) {
  return AccType(1) / sqrt(v);
}

template<>
__device__ __forceinline__ float rsqrt<float>(float v) {
  return rsqrtf(v);
}

template<>
__device__ __forceinline__ double rsqrt<double>(double v) {
  return rsqrt(v);
}

// shared memory struct
namespace {
template <typename T>
struct SharedMemory;

template <>
struct SharedMemory<float> {
    __device__ float* getPointer() {
        extern __shared__ float s_float[];
        return s_float;
    }
};

template <>
struct SharedMemory<double> {
    __device__ double* getPointer() {
        extern __shared__ double s_double[];
        return s_double;
    }
};
}

template<typename InputType, typename AccType, typename OutputType>
__device__ void cuApplyRMSNorm_(
  OutputType* __restrict__ output_vals,
  AccType* __restrict__ invvar,
  const InputType* __restrict__ vals,
  const int n1,
  const int n2,
  const AccType epsilon,
  const OutputType* __restrict__ gamma)
{
  // Assumptions:
  // 1) blockDim.x == warpSize
  // 2) Tensors are contiguous
  //
  for (auto i1 = blockIdx.y; i1 < n1; i1 += gridDim.y) {
    SharedMemory<AccType> shared;
    AccType* buf = shared.getPointer();
    AccType sigma2;
    cuWelfordSigma2(vals, n1, n2, i1, sigma2, buf);

    const InputType* lvals = vals + i1*n2;
    OutputType* ovals = output_vals + i1*n2;
    AccType c_invvar = rsqrt(sigma2 + epsilon);
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    if (gamma != NULL) {
      for (int i = thrx;  i < n2;  i += numx) {
        AccType curr = static_cast<AccType>(__ldg(&lvals[i]));
        ovals[i] = gamma[i] * static_cast<OutputType>(c_invvar * curr);
      }
    } else {
      for (int i = thrx;  i < n2;  i += numx) {
        AccType curr = static_cast<AccType>(__ldg(&lvals[i]));
        ovals[i] = static_cast<OutputType>(c_invvar * curr);
      }
    }
    if (threadIdx.x == 0 && threadIdx.y == 0) {
      invvar[i1] = c_invvar;
    }
    __syncthreads();
  }
}

template<typename InputType, typename AccType, typename OutputType=InputType>
__global__ void cuApplyRMSNorm(
  OutputType* __restrict__ output_vals,
  AccType* __restrict__ invvar,
  const InputType* __restrict__ vals,
  const int n1,
  const int n2,
  const AccType epsilon,
  const OutputType* __restrict__ gamma)
{
  cuApplyRMSNorm_<InputType, AccType, OutputType>(output_vals, invvar, vals, n1, n2, epsilon, gamma);
}

template<typename OutputType>
__device__ OutputType clamp_by_magnitude(OutputType curr_gamma, double eps)
{
  const OutputType kMinGamma = OutputType(eps);
  return abs(curr_gamma) < kMinGamma ? (curr_gamma < 0 ? -kMinGamma : kMinGamma) : curr_gamma;
}


template<typename InputType, typename AccType, typename OutputType, bool MemoryEfficient>
__device__ void cuLoadAddStridedInputsRMSOnly(
    const int i1_block, const int thr_load_row_off, const int thr_load_col_off,
    const int i2_off, const int row_stride, AccType* warp_buf2,
    const InputType* input_or_output, const OutputType* dout, const int i1_end, const int n2,
    const AccType* __restrict__ invvar, const OutputType* __restrict__ gamma, const double eps) {
    int i1 = i1_block + thr_load_row_off;
    if (i1 < i1_end) {
        for (int k = 0; k < blockDim.y; ++k) {
            int i2 = i2_off + k;
            int load_idx = i1 * n2 + i2;
            int write_idx = thr_load_row_off * row_stride + thr_load_col_off + k;
            if (i2 < n2) {
                AccType c_h = static_cast<AccType>(__ldg(&input_or_output[load_idx]));
                AccType curr_dout = static_cast<AccType>(__ldg(&dout[load_idx]));
                if (MemoryEfficient) {
                    warp_buf2[write_idx] += curr_dout * c_h / static_cast<AccType>(clamp_by_magnitude(gamma[i2], eps));
                } else {
                    warp_buf2[write_idx] += curr_dout * c_h * invvar[i1];
                }
            }
        }
    }
}

template<typename InputType, typename AccType, typename OutputType, bool MemoryEfficient>
__device__ void cuLoadWriteStridedInputsRMSOnly(
    const int i1_block, const int thr_load_row_off, const int thr_load_col_off,
    const int i2_off, const int row_stride, AccType* warp_buf2,
    const InputType* input_or_output, const OutputType* dout, const int i1_end, const int n2,
    const AccType* __restrict__ invvar, const OutputType* __restrict__ gamma, const double eps) {
    int i1 = i1_block + thr_load_row_off;
    if (i1 < i1_end) {
        for (int k = 0; k < blockDim.y; ++k) {
            int i2 = i2_off + k;
            int load_idx = i1 * n2 + i2;
            int write_idx = thr_load_row_off * row_stride + thr_load_col_off + k;
            if (i2 < n2) {
                AccType c_h = static_cast<AccType>(__ldg(&input_or_output[load_idx]));
                AccType curr_dout = static_cast<AccType>(__ldg(&dout[load_idx]));
                if (MemoryEfficient) {
                    warp_buf2[write_idx] = curr_dout * c_h / static_cast<AccType>(clamp_by_magnitude(gamma[i2], eps));
                } else {
                    warp_buf2[write_idx] = curr_dout * c_h * invvar[i1];
                }
            } else {
                warp_buf2[write_idx] = AccType(0);
            }
        }
    } else {
        for (int k = 0; k < blockDim.y; ++k) {
            int write_idx = thr_load_row_off * row_stride + thr_load_col_off + k;
            warp_buf2[write_idx] = AccType(0);
        }
    }
}
template<typename InputType, typename AccType, typename OutputType, bool MemoryEfficient>
__global__ void cuComputeGradInput(
    const OutputType* __restrict__ dout,
    const InputType* __restrict__ input_or_output,
    const int n1,
    const int n2,
    const AccType* __restrict__ invvar,
    AccType epsilon,
    const OutputType* gamma,
    InputType* grad_input,
    const double eps)
{
    for (auto i1 = blockIdx.y; i1 < n1; i1 += gridDim.y) {
        AccType sum_loss2 = AccType(0);
        const InputType* k_h = input_or_output + i1*n2;
        const OutputType* k_dout = dout + i1*n2;
        const AccType c_invvar = invvar[i1];
        const int numx = blockDim.x * blockDim.y;
        const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
        if (gamma != NULL) {
            int l = 4*thrx;
            for (;  l+3 < n2;  l+=4*numx) {
                #pragma unroll
                for (int k = 0;  k < 4;  ++k) {
                    const AccType c_h = static_cast<AccType>(__ldg(&k_h[l+k]));
                    const AccType c_loss = static_cast<AccType>(__ldg(&k_dout[l+k]));
                    if (MemoryEfficient) {
                        sum_loss2 += c_loss * c_h;
                    } else {
                        sum_loss2 += c_loss * gamma[l+k] * c_h * c_invvar;
                    }
                }
            }
            for (;  l < n2;  ++l) {
                const AccType c_h = static_cast<AccType>(__ldg(&k_h[l]));
                const AccType c_loss = static_cast<AccType>(__ldg(&k_dout[l]));
                if (MemoryEfficient) {
                    sum_loss2 += c_loss * c_h;
                } else {
                    sum_loss2 += c_loss * gamma[l] * c_h * c_invvar;
                }
            }
        } else {
            int l = 4*thrx;
            for (;  l+3 < n2;  l+=4*numx) {
                #pragma unroll
                for (int k = 0;  k < 4;  ++k) {
                    const AccType c_h = static_cast<AccType>(__ldg(&k_h[l+k]));
                    const AccType c_loss = static_cast<AccType>(__ldg(&k_dout[l+k]));
                    if (MemoryEfficient) {
                        sum_loss2 += c_loss * c_h;
                    } else {
                        sum_loss2 += c_loss * c_h * c_invvar;
                    }
                }
            }
            for (;  l < n2;  ++l) {
                const AccType c_h = static_cast<AccType>(__ldg(&k_h[l]));
                const AccType c_loss = static_cast<AccType>(__ldg(&k_dout[l]));
                if (MemoryEfficient) {
                    sum_loss2 += c_loss * c_h;
                } else {
                    sum_loss2 += c_loss * c_h * c_invvar;
                }
            }
        }
        // intra-warp reductions
        #pragma unroll
        for (int mask = blockDim.x/2;  mask > 0;  mask /= 2) {
            sum_loss2 += WARP_SHFL_XOR(sum_loss2, mask);
        }
        // inter-warp reductions
        if (blockDim.y > 1) {
            SharedMemory<AccType> shared;
            AccType* buf = shared.getPointer();
            for (int offset = blockDim.y/2;  offset > 0;  offset /= 2) {
                // upper half of warps write to shared
                if (threadIdx.y >= offset && threadIdx.y < 2*offset) {
                    const int wrt_i = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
                    buf[wrt_i] = sum_loss2;
                }
                __syncthreads();
                // lower half merges
                if (threadIdx.y < offset) {
                    const int read_i = threadIdx.y * blockDim.x + threadIdx.x;
                    sum_loss2 += buf[read_i];
                }
                __syncthreads();
            }
            if (threadIdx.y == 0) {
                buf[threadIdx.x] = sum_loss2;
            }
            __syncthreads();
            if (threadIdx.y != 0) {
                sum_loss2 = buf[threadIdx.x];
            }
        }
        // all threads now have the sum over l
        const AccType fH = static_cast<AccType>(n2);
        const AccType term1 = (AccType(1) / fH) * c_invvar;
        InputType* k_grad_input = grad_input + i1*n2;
        if (gamma != NULL) {
            for (int l = thrx;  l < n2;  l+=numx) {
                const AccType c_h = static_cast<AccType>(__ldg(&k_h[l]));
                const AccType c_loss = static_cast<AccType>(__ldg(&k_dout[l]));
                const AccType k_gamma = static_cast<AccType>(clamp_by_magnitude(gamma[l], eps));
                AccType f_grad_input = fH * c_loss * k_gamma;
                if (MemoryEfficient) {
                    f_grad_input -= c_h / k_gamma * sum_loss2;
                } else {
                    f_grad_input -= c_h * c_invvar * sum_loss2;
                }
                f_grad_input *= term1;
                k_grad_input[l] = static_cast<InputType>(f_grad_input);
            }
        } else {
            for (int l = thrx;  l < n2;  l+=numx) {
                const AccType c_h = static_cast<AccType>(__ldg(&k_h[l]));
                const AccType c_loss = static_cast<AccType>(__ldg(&k_dout[l]));
                AccType f_grad_input = fH * c_loss;
                if (MemoryEfficient) {
                    f_grad_input -= c_h * sum_loss2;
                } else {
                    f_grad_input -= c_h * c_invvar * sum_loss2;
                }
                f_grad_input *= term1;
                k_grad_input[l] = static_cast<InputType>(f_grad_input);
            }
        }
        // prevent race where buf is written again before reads are done
        __syncthreads();
    }
}
template<typename GradType, typename SumType>
__global__ void cuComputeGradGamma(
    const GradType* __restrict__ part_grad_gamma,
    const int part_size,
    const int n1,
    const int n2,
    SumType* __restrict__ grad_gamma)
{
    // sum partial gradients for gamma
    SharedMemory<GradType> shared;
    GradType* buf = shared.getPointer();
    const int i2 = blockIdx.x * blockDim.x + threadIdx.x;
    if (i2 < n2) {
        // each warp does sequential reductions until reduced part_size is num_warps
        const int num_warp_reductions = part_size / blockDim.y;
        GradType sum_gamma = GradType(0);
        const GradType* part_grad_gamma_ptr = part_grad_gamma + threadIdx.y * num_warp_reductions * n2 + i2;
        for (int warp_offset = 0; warp_offset < num_warp_reductions; ++warp_offset) {
            sum_gamma += static_cast<GradType>(__ldg(&part_grad_gamma_ptr[warp_offset*n2]));
        }
        // inter-warp reductions
        for (int offset = blockDim.y/2; offset >= 1; offset /= 2) {
            // top half write to shared memory
            if (threadIdx.y >= offset && threadIdx.y < 2*offset) {
                const int write_idx = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
                buf[write_idx] = sum_gamma;
            }
            __syncthreads();
            // bottom half sums
            if (threadIdx.y < offset) {
                const int read_idx = threadIdx.y * blockDim.x + threadIdx.x;
                sum_gamma += buf[read_idx];
            }
            __syncthreads();
        }
        // write out fully summed gradients
        if (threadIdx.y == 0) {
            grad_gamma[i2] = sum_gamma;
        }
    }
}
template<typename InputType, typename AccType, typename OutputType, bool MemoryEfficient>
__global__ void cuComputePartGradGammaBeta(
    const OutputType* __restrict__ dout,
    const InputType* __restrict__ input_or_output,
    const int n1,
    const int n2,
    const AccType* __restrict__ invvar,
    const OutputType* __restrict__ gamma,
    const double eps,
    AccType* __restrict__ part_grad_gamma)
{
    const int blockSize = blockDim.y * blockDim.y;
    const int numsegs_n1 = (n1 + blockSize - 1) / blockSize;
    const int segs_per_block = (numsegs_n1 + gridDim.y - 1) / gridDim.y;
    const int i1_beg = blockIdx.y * segs_per_block * blockSize;
    const int i1_beg_plus_one = (blockIdx.y + 1) * segs_per_block * blockSize;
    const int i1_end = std::min(i1_beg_plus_one, n1);
    const int row_stride = blockDim.x + 1;
    const int thr_load_col_off = (threadIdx.x * blockDim.y) & (blockDim.x - 1);
    const int thr_load_row_off = (threadIdx.x * blockDim.y) / blockDim.x + threadIdx.y * blockDim.y;
    const int i2_off = blockIdx.x * blockDim.x + thr_load_col_off;

    SharedMemory<AccType> shared;
    AccType* buf = shared.getPointer();
    AccType* warp_buf2 = buf;

    cuLoadWriteStridedInputsRMSOnly<InputType, AccType, OutputType, MemoryEfficient>(
        i1_beg, thr_load_row_off, thr_load_col_off, i2_off, row_stride,
        warp_buf2, input_or_output, dout, i1_end, n2, invvar, gamma, eps);

    for (int i1_block = i1_beg + blockSize; i1_block < i1_end; i1_block += blockSize) {
        cuLoadAddStridedInputsRMSOnly<InputType, AccType, OutputType, MemoryEfficient>(
            i1_block, thr_load_row_off, thr_load_col_off, i2_off, row_stride,
            warp_buf2, input_or_output, dout, i1_end, n2, invvar, gamma, eps);
    }
    __syncthreads();

    // inter-warp reductions
    // sum within each warp
    AccType acc2 = AccType(0);
    for (int k = 0; k < blockDim.y; ++k) {
        int row1 = threadIdx.y + k * blockDim.y;
        int idx1 = row1 * row_stride + threadIdx.x;
        acc2 += warp_buf2[idx1];
    }
    warp_buf2[threadIdx.y * row_stride + threadIdx.x] = acc2;
    __syncthreads();

    // sum all warps
    #pragma unroll
    for (int offset = blockDim.y / 2; offset > 1; offset /= 2) {
        if (threadIdx.y < offset) {
            int row1 = threadIdx.y;
            int row2 = threadIdx.y + offset;
            int idx1 = row1 * row_stride + threadIdx.x;
            int idx2 = row2 * row_stride + threadIdx.x;
            warp_buf2[idx1] += warp_buf2[idx2];
        }
        __syncthreads();
    }

    const int i2 = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIdx.y == 0 && i2 < n2) {
        int row1 = threadIdx.y;
        int row2 = threadIdx.y + 1;
        int idx1 = row1 * row_stride + threadIdx.x;
        int idx2 = row2 * row_stride + threadIdx.x;
        part_grad_gamma[blockIdx.y * n2 + i2] = warp_buf2[idx1] + warp_buf2[idx2];
    }
}

template<typename InputType, typename AccType, typename OutputType>
void HostRMSNormGradient(
    const OutputType* dout,
    const AccType* invvar,
    at::Tensor* input_or_output,
    int n1,
    int n2,
    const OutputType* gamma,
    double epsilon,
    InputType* grad_input,
    OutputType* grad_gamma,
    bool memory_efficient)
{
    auto stream = at::cuda::getCurrentCUDAStream().stream();

    if (gamma != nullptr) {
        constexpr int part_size = 16;
        constexpr int threads_x = 32;
        constexpr int threads_y = 4;
        const dim3 threads2(threads_x, threads_y, 1);
        const dim3 blocks2((n2 + threads_x - 1) / threads_x, part_size, 1);
        constexpr int nshared2_a = 2 * sizeof(AccType) * threads_y * threads_y * (threads_x + 1);
        constexpr int nshared2_b = threads_x * threads_y * sizeof(AccType);
        constexpr int nshared2 = nshared2_a > nshared2_b ? nshared2_a : nshared2_b;

        const auto part_grad_dtype = (input_or_output->scalar_type() == at::ScalarType::Half ||
                                      input_or_output->scalar_type() == at::ScalarType::BFloat16)
                                     ? at::ScalarType::Float
                                     : input_or_output->scalar_type();

        at::Tensor part_grad_gamma = at::empty({part_size, n2}, input_or_output->options().dtype(part_grad_dtype));

        BOOL_SWITCH(memory_efficient, MemoryEfficient, [&] {
            auto kernel = &cuComputePartGradGammaBeta<InputType, AccType, OutputType, MemoryEfficient>;
            kernel<<<blocks2, threads2, nshared2, stream>>>(
                dout,
                input_or_output->DATA_PTR<InputType>(),
                n1, n2,
                invvar,
                gamma,
                epsilon,
                part_grad_gamma.DATA_PTR<AccType>());
        });

        constexpr int threads3_x = 32;
        constexpr int threads3_y = 8;
        const dim3 threads3(threads3_x, threads3_y, 1);
        const dim3 blocks3((n2 + threads3_x - 1) / threads3_x, 1, 1);
        constexpr int nshared3 = threads3_x * threads3_y * sizeof(AccType);

        cuComputeGradGamma<<<blocks3, threads3, nshared3, stream>>>(
            part_grad_gamma.DATA_PTR<AccType>(),
            part_size,
            n1, n2,
            grad_gamma);
    }

    // compute grad_input
    const uint64_t maxGridY = at::cuda::getCurrentDeviceProperties()->maxGridSize[1];
    const dim3 blocks1(1, std::min((uint64_t)n1, maxGridY), 1);
    constexpr int threads1_x = 32;
    constexpr int threads1_y = 4;
    const dim3 threads1(threads1_x, threads1_y, 1);
    constexpr int nshared = threads1_y > 1 ? threads1_y * threads1_x * sizeof(AccType) : 0;

    BOOL_SWITCH(memory_efficient, MemoryEfficient, [&] {
        auto kernel = &cuComputeGradInput<InputType, AccType, OutputType, MemoryEfficient>;
        kernel<<<blocks1, threads1, nshared, stream>>>(
            dout,
            input_or_output->DATA_PTR<InputType>(),
            n1, n2,
            invvar,
            epsilon,
            gamma,
            grad_input,
            epsilon);
    });
}
void cuda_rms_norm_gradient(
    at::Tensor* dout,
    at::Tensor* invvar,
    at::Tensor* input_or_output,
    int n1,
    int n2,
    at::IntArrayRef normalized_shape,
    at::Tensor* gamma,
    double epsilon,
    at::Tensor* grad_input,
    at::Tensor* grad_gamma,
    bool memory_efficient)
{
    using namespace at;
    DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT_INOUT_TYPES(
        input_or_output->scalar_type(), gamma == nullptr ? input_or_output->scalar_type() : gamma->scalar_type(),
        "cuComputeGradInputRMS", [&] {
            using accscalar_t = at::acc_type<scalar_t_in, true>;
            HostRMSNormGradient(
                dout->DATA_PTR<scalar_t_out>(),
                invvar->DATA_PTR<accscalar_t>(),
                input_or_output,
                n1, n2,
                gamma != nullptr ? gamma->DATA_PTR<scalar_t_out>() : nullptr,
                epsilon,
                grad_input->DATA_PTR<scalar_t_in>(),
                gamma != nullptr ? grad_gamma->DATA_PTR<scalar_t_out>() : nullptr,
                memory_efficient);
        }
    );
}

template<typename InputType, typename AccType, typename OutputType>
void HostApplyRMSNorm(
    OutputType* output,
    AccType* invvar,
    const InputType* input,
    int n1,
    int n2,
    double epsilon,
    const OutputType* gamma)
{
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    constexpr int threads_x = 32;
    constexpr int threads_y = 4;
    const dim3 threads(threads_x, threads_y, 1);
    const uint64_t maxGridY = at::cuda::getCurrentDeviceProperties()->maxGridSize[1];
    const dim3 blocks(1, std::min((uint64_t)n1, maxGridY), 1);
    constexpr int nshared = threads_y > 1 ? threads_y * sizeof(AccType) + (threads_y / 2) * sizeof(AccType) : 0;
    cuApplyRMSNorm<InputType, AccType, OutputType><<<blocks, threads, nshared, stream>>>(
        output, invvar, input, n1, n2, AccType(epsilon), gamma);
}

void cuda_rms_norm(
    at::Tensor* output,
    at::Tensor* invvar,
    at::Tensor* input,
    int n1,
    int n2,
    at::IntArrayRef normalized_shape,
    at::Tensor* gamma,
    double epsilon)
{
    using namespace at;
    DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT_INOUT_TYPES(
        input->scalar_type(), output->scalar_type(), "rms_norm_cuda_kernel", [&] {
            using accscalar_t = acc_type<scalar_t_in, true>;
            HostApplyRMSNorm<scalar_t_in, accscalar_t, scalar_t_out>(
                output->DATA_PTR<scalar_t_out>(),
                invvar->DATA_PTR<accscalar_t>(),
                input->DATA_PTR<scalar_t_in>(),
                n1, n2,
                epsilon,
                gamma != nullptr ? gamma->DATA_PTR<scalar_t_out>() : nullptr);
        }
    );
}
